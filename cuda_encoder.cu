#include "hip/hip_runtime.h"
//
// Created by yejiongtao on 2019/5/2.
//

#include "cuda_encoder.h"
#include "galois.h"
#include "cycletimer.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>


const int BYTES_PER_THREAD = 1;
const int THREADS_PER_BLOCK = 128;
char *cuda_parity_rows;
char *cuda_mult_table;

//#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans)  cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr, "CUDA Error: %s at %s:%d\n",
        hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#else
#define cudaCheckError(ans) ans
#endif

__inline__ int updiv(int a, int b) {
    return (a + b - 1) / b;
}

void init_cuda(char *parity_rows, int r, int c) {
    cudaCheckError(hipMalloc(&cuda_parity_rows, r * c * sizeof(char)));
    cudaCheckError(hipMemcpy(cuda_parity_rows, parity_rows,
            r * c * sizeof(char), hipMemcpyHostToDevice));
}

void copy_mult_table_to_cuda() {
    cudaCheckError(hipMalloc(&cuda_mult_table, FIELD_SIZE * FIELD_SIZE * sizeof(char)));
    // because MULTIPLICATION_TABLE is a 2D array
    for(int i = 0; i < FIELD_SIZE; i++)
        cudaCheckError(hipMemcpy(cuda_mult_table + i * FIELD_SIZE, MULTIPLICATION_TABLE[i],
                FIELD_SIZE * sizeof(char), hipMemcpyHostToDevice));
}

__global__ void kernel_encode(char *matrix_rows, int matrix_r, int matrix_c,
                              char *inputs, int input_r, int input_c,
                              char *outputs, int output_r, int output_c,
                              int offset, int byte_count, char *mult_table) {
    int i_thread = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i_output = 0; i_output < output_r; i_output++) {
        char *output_shard = outputs + i_output * output_c;
        char *matrix_row = matrix_rows + i_output * matrix_c;
        {
            int i_input = 0;
            char *input_shard = inputs + i_input * input_c;
            char *mult_table_row = mult_table + (matrix_row[i_input] & 0xFF) * FIELD_SIZE;
            for(int i_byte = i_thread * BYTES_PER_THREAD;
                        i_byte < (i_thread+1) * BYTES_PER_THREAD; i_byte++) {
                if (i_byte >= input_r)
                    break;
                output_shard[i_byte] = mult_table_row[input_shard[i_byte] & 0xFF];
            }
        }
        for (int i_input = 1; i_input < input_r; i_input++) {
            char *input_shard = inputs + i_input * input_c;
            char *mult_table_row = mult_table + (matrix_row[i_input] & 0xFF) * FIELD_SIZE;
            for(int i_byte = i_thread * BYTES_PER_THREAD;
                i_byte < (i_thread+1) * BYTES_PER_THREAD; i_byte++) {
                if (i_byte >= input_r)
                    break;
                output_shard[i_byte] ^= mult_table_row[input_shard[i_byte] & 0xFF];
            }
        }
    }
}

void output_input_byte_table_cuda_loop(char *matrix_rows, int matrix_r, int matrix_c,
                                       char *inputs, int input_r, int input_c,
                                       char *outputs, int output_r, int output_c,
                                       int offset, int byte_count) {
    int num_blocks = updiv(updiv(byte_count, BYTES_PER_THREAD), THREADS_PER_BLOCK);

    char *cuda_inputs;
    cudaCheckError(hipMalloc(&cuda_inputs,
            (input_c * input_r + output_c * output_r) * sizeof(char)));
    char *cuda_outputs = cuda_inputs + input_c * input_r;
    cudaCheckError(hipMemcpy(cuda_inputs, inputs,
            input_c * input_r * sizeof(char), hipMemcpyHostToDevice));

    kernel_encode<<<num_blocks, THREADS_PER_BLOCK>>>(
            cuda_parity_rows, matrix_r, matrix_c, cuda_inputs, input_r, input_c,
            cuda_outputs, output_r, output_c, offset, byte_count, cuda_mult_table);
    cudaCheckError(hipDeviceSynchronize());

    cudaCheckError(hipMemcpy(outputs, cuda_outputs,
            output_c * output_r * sizeof(char), hipMemcpyDeviceToHost));
    hipFree(cuda_inputs);
}
